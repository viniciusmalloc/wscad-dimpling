#include <iostream>
#include <iomanip>
#include <algorithm>
#include <vector>
#include <cstdio>
#include <cstdlib>
#include <cmath>
#include <hip/hip_runtime.h>
#include <omp.h>

#define C 4
#define THREADS 1024 // 2^10
#define MAX 100
#define MAXS MAX*MAX
#define PERM_MAX (MAX*(MAX-1)*(MAX-2)*(MAX-3))/24

#define gpuErrChk(ans){ gpuAssert((ans), __FILE__, __LINE__); }
inline void gpuAssert(hipError_t code, char *file, int line, bool abort = true)
{
    if (code != hipSuccess)
    {
        fprintf(stderr, "GPUassert: %s %s %d\n", hipGetErrorString(code),
            file, line);
        if (abort) getchar();
    }
}

using namespace std;

/*
    sz          ---> Adjacency matrix dimension (1D)
    perm        ---> Number of permutations of an instance
    graph       ---> Adjacency matrix itself
    seeds       ---> Set of seeds
    faces       ---> Set of triangular faces for the output
*/
struct Node
{
    int sz, perm;
    int graph[MAXS], seeds[C*PERM_MAX], F_ANS[6*MAX];
};

/*
    faces       ---> Number of triangular faces
    count       ---> Number of remaining vertices
    tmpMax      ---> Max value obtained for a seed
    F           ---> Set of triangular faces
    F           ---> Set of remaining vertices
*/
struct Params
{
    int *faces, *count, *tmpMax;
    int *F, *V;
};

/*
    SIZE        ---> Number of vertices
    BLOCKS      ---> Number of blocks
    PERM        ---> Number of permutations
    R           ---> Output graph for a possible solution
    F           ---> Set of triangular faces of an instance
    qtd         ---> Number of possible 4-cliques
*/
int SIZE, PERM, GPU_CNT = 1;
int R[MAXS], F[8 * MAX], bib[MAX];
int qtd = 0;

Node *N;

//-----------------------------------------------------------------------------
// Mac OSX
#ifdef __MACH__
#include <mach/clock.h>
#include <mach/mach.h>
#endif

/*
    Prints elapsed time.
    */
void printElapsedTime(double start, double stop)
{
    double elapsed = stop - start;
    printf("Elapsed time: %.3lfs.\n", elapsed);
}
//-----------------------------------------------------------------------------
/*  
    Gets clock time.
    */
void current_utc_time(struct timespec *ts) 
{
    #ifdef __MACH__ // OS X does not have clock_gettime, use clock_get_time
        clock_serv_t cclock;
        mach_timespec_t mts;
        host_get_clock_service(mach_host_self(), CALENDAR_CLOCK, &cclock);
        clock_get_time(cclock, &mts);
        mach_port_deallocate(mach_task_self(), cclock);
        ts->tv_sec = mts.tv_sec;
        ts->tv_nsec = mts.tv_nsec;
    #else
        clock_gettime(CLOCK_REALTIME, ts);
    #endif
}
//-----------------------------------------------------------------------------
double getTime()
{
    timespec ts;
    current_utc_time(&ts);
    return double(ts.tv_sec) + double(ts.tv_nsec) / 1e9;
}
//-----------------------------------------------------------------------------
/*
    t   ---> thread index
    Generates a list of vertices which are not on the initial planar graph.
    */
__device__ void generateVertexList(Node* devN, Params* devP, int t, int offset)
{
    int sz = devN->sz, perm = devN->perm;

    int va = devN->seeds[(t + offset) * 4],
        vb = devN->seeds[(t + offset) * 4 + 1],
        vc = devN->seeds[(t + offset) * 4 + 2],
        vd = devN->seeds[(t + offset) * 4 + 3];
    for (int i = 0; i < sz; ++i)
    {
        if (i == va || i == vb || i == vc || i == vd) devP->V[t + i * perm] = -1;
        else devP->V[t + i * perm] = i;
    }
}
//-----------------------------------------------------------------------------
/*
    Returns the weight of the planar graph so far
*/
__device__ void generateFaceList(Node* devN, Params* devP, int graph[], int t,
    int offset)
{
    int sz = devN->sz, perm = devN->perm;

    int va = devN->seeds[(t + offset) * 4],
        vb = devN->seeds[(t + offset) * 4 + 1],
        vc = devN->seeds[(t + offset) * 4 + 2],
        vd = devN->seeds[(t + offset) * 4 + 3];

    // Generate the first triangle of the output graph
    devP->F[t + (devP->faces[t] * 3) * perm] = va;
    devP->F[t + (devP->faces[t] * 3 + 1) * perm] = vb;
    devP->F[t + ((devP->faces[t]++) * 3 + 2) * perm] = vc;

    // Generate the next 3 possible faces
    devP->F[t + (devP->faces[t] * 3) * perm] = va;
    devP->F[t + (devP->faces[t] * 3 + 1) * perm] = vb;
    devP->F[t + ((devP->faces[t]++) * 3 + 2) * perm] = vd;

    devP->F[t + (devP->faces[t] * 3) * perm] = va;
    devP->F[t + (devP->faces[t] * 3 + 1) * perm] = vc;
    devP->F[t + ((devP->faces[t]++) * 3 + 2) * perm] = vd;

    devP->F[t + (devP->faces[t] * 3) * perm] = vb;
    devP->F[t + (devP->faces[t] * 3 + 1) * perm] = vc;
    devP->F[t + ((devP->faces[t]++) * 3 + 2) * perm] = vd;

    int resp = graph[va*sz + vb] + graph[va*sz + vc] + graph[vb*sz + vc];
    resp += graph[va*sz + vd] + graph[vb*sz + vd] + graph[vc*sz + vd];
    devP->tmpMax[t] = resp;
}
//-----------------------------------------------------------------------------
/*
    Inserts a new vertex, 3 new triangular faces
    and removes the face from the list.
    */
__device__ int faceDimple(Node* devN, Params* devP, int graph[], int new_vertex,
    int f, int t)
{
    int sz = devN->sz, perm = devN->perm;

    // Remove the chosen face and insert a new one
    int va = devP->F[t + (f * 3) * perm],
        vb = devP->F[t + (f * 3 + 1) * perm],
        vc = devP->F[t + (f * 3 + 2) * perm];

    devP->F[t + (f * 3) * perm] = new_vertex,
    devP->F[t + (f * 3 + 1) * perm] = va,
    devP->F[t + (f * 3 + 2) * perm] = vb;
    
    // Insert the other two possible faces
    devP->F[t + (devP->faces[t] * 3) * perm] = new_vertex;
    devP->F[t + (devP->faces[t] * 3 + 1) * perm] = va;
    devP->F[t + ((devP->faces[t]++) * 3 + 2) * perm] = vc;

    devP->F[t + (devP->faces[t] * 3) * perm] = new_vertex;
    devP->F[t + (devP->faces[t] * 3 + 1) * perm] = vb;
    devP->F[t + ((devP->faces[t]++) * 3 + 2) * perm] = vc;

    int resp = graph[va*sz + new_vertex] + graph[vb*sz + new_vertex]
        + graph[vc*sz + new_vertex];

    return resp;
}
//-----------------------------------------------------------------------------
/*
    Returns the vertex having the maximum gain
    inserting within a face.
    */
__device__ int maxGainFace(Node* devN, Params* devP, int graph[], int* f, int t)
{
    int sz = devN->sz, perm = devN->perm;
    int gain = -1, vertex = -1;

    // Iterate through the remaining vertices
    for (int new_vertex = 0; new_vertex < sz; ++new_vertex)
    {
        if (devP->V[t + new_vertex * perm] == -1) continue;
        // Test the dimple on each face
        int faces = devP->faces[t];
        for (int i = 0; i < faces; ++i)
        {
            int va = devP->F[t + (i * 3) * perm],
                vb = devP->F[t + (i * 3 + 1) * perm],
                vc = devP->F[t + (i * 3 + 2) * perm];
            int tmpGain = graph[va*sz + new_vertex] + graph[vb*sz + new_vertex]
                + graph[vc*sz + new_vertex];
            if (tmpGain > gain)
            {
                gain = tmpGain;
                *f = i;
                vertex = new_vertex;
            }
        }
    }
    return vertex;
}
//-----------------------------------------------------------------------------
__device__ void dimpling(Node* devN, Params* devP, int graph[], int t)
{
    int perm = devN->perm;
    while (devP->count[t])
    {
        int f = -1;
        int vertex = maxGainFace(devN, devP, graph, &f, t);
        devP->V[t + vertex * perm] = -1;
        devP->tmpMax[t] += faceDimple(devN, devP, graph, vertex, f, t);
        devP->count[t]--;
    }
}
//-----------------------------------------------------------------------------
__device__ void copyGraph(Node *devN, Params *devP, int t)
{
    int faces = devP->faces[t], perm = devN->perm;
    for (int i = 0; i < faces; ++i)
    {
        int va = devP->F[t + (i * 3) * perm],
            vb = devP->F[t + (i * 3 + 1) * perm],
            vc = devP->F[t + (i * 3 + 2) * perm];
        devN->F_ANS[i * 3] = va,
        devN->F_ANS[i * 3 + 1] = vb,
        devN->F_ANS[i * 3 + 2] = vc;
    }
}
//-----------------------------------------------------------------------------
__device__ void initializeDevice(Params *devP, int sz, int t)
{
    devP->faces[t] = 0;
    devP->tmpMax[t] = -1;
    devP->count[t] = sz - 4;
}
//-----------------------------------------------------------------------------
__global__ void solve(Node *devN, Params devP, int *respMax, int offset)
{
    int x = blockDim.x*blockIdx.x + threadIdx.x;
    int sz = devN->sz, perm = devN->perm;
    extern __shared__ int graph[];

    for (int i = threadIdx.x; i < sz*sz; i += blockDim.x)
        graph[i] = devN->graph[i];
    __syncthreads();

    if (x < perm)
    {
        initializeDevice(&devP, sz, x);
        generateVertexList(devN, &devP, x, offset);
        generateFaceList(devN, &devP, graph, x, offset);
        dimpling(devN, &devP, graph, x);
        atomicMax(respMax, devP.tmpMax[x]);
        __syncthreads();

        if (devP.tmpMax[x] == *respMax)
            copyGraph(devN, &devP, x);
        __syncthreads();
    }
}
//-----------------------------------------------------------------------------
int prepare()
{
    int finalResp = -1, pos = -1;

    #pragma omp parallel for num_threads(GPU_CNT)
    for (int gpu_id = 0; gpu_id < GPU_CNT; gpu_id++)
    {
        hipSetDevice(gpu_id);
        int range = (int)ceil(PERM / (double)GPU_CNT);
        int perm = ((gpu_id + 1)*range > PERM ? PERM - gpu_id*range : range);
        int offset = gpu_id*range;
        N->perm = perm;

        int resp = -1, *tmpResp;
        gpuErrChk(hipMalloc((void**)&tmpResp, sizeof(int)));
        gpuErrChk(hipMemcpy(tmpResp, &resp, sizeof(int), hipMemcpyHostToDevice));

        Node *devN;
        Params devP;

        size_t sz = 3*range*sizeof(int) + SIZE*range*sizeof(int)
            + (6*SIZE)*range*sizeof(int);

        printf("Using %d MBytes in Kernel %d\n", (sz + sizeof(Node)) / (1 << 20), gpu_id);

        gpuErrChk(hipMalloc((void**)&devN, sizeof(Node)));
        gpuErrChk(hipMemcpy(devN, N, sizeof(Node), hipMemcpyHostToDevice));

        gpuErrChk(hipMalloc((void**)&devP.faces, perm*sizeof(int)));
        gpuErrChk(hipMalloc((void**)&devP.count, perm*sizeof(int)));
        gpuErrChk(hipMalloc((void**)&devP.tmpMax, perm*sizeof(int)));
        gpuErrChk(hipMalloc((void**)&devP.F, 6*SIZE*perm*sizeof(int)));
        gpuErrChk(hipMalloc((void**)&devP.V, SIZE*perm*sizeof(int)));

        dim3 blocks(perm/THREADS + 1, 1);
        dim3 threads(THREADS, 1);

        printf("Kernel %d launched with %d blocks, each w/ %d threads\n",
            gpu_id, range/THREADS + 1, THREADS);
        solve <<<blocks, threads, SIZE*SIZE*sizeof(int)>>>(devN, devP, tmpResp, offset);
        gpuErrChk(hipDeviceSynchronize());

        // Copy back the maximum weight and the set of faces
        // which gave this result
        gpuErrChk(hipMemcpy(&resp, tmpResp, sizeof(int), hipMemcpyDeviceToHost));
        printf("Kernel finished.\nLocal maximum found in Kernel %d: %d\n",
            gpu_id+1, resp);
        printf("Copying results...\n");

        #pragma omp barrier
        {
            #pragma omp critical
            {
                if (resp > finalResp)
                {
                    finalResp = resp;
                    pos = gpu_id;
                }
            }
        }

        if (pos == gpu_id)
        {
            gpuErrChk(hipMemcpy(&F, devN->F_ANS, 6*MAX*sizeof(int),
                hipMemcpyDeviceToHost));
        }

        printf("Freeing memory...\n");
        gpuErrChk(hipFree(devN));
        gpuErrChk(hipFree(devP.faces));
        gpuErrChk(hipFree(devP.count));
        gpuErrChk(hipFree(devP.tmpMax));
        gpuErrChk(hipFree(devP.F));
        gpuErrChk(hipFree(devP.V));
        gpuErrChk(hipDeviceReset());
    }

    return finalResp;
}
//-----------------------------------------------------------------------------
/*
    C      ---> Size of the combination
    index  ---> Current index in data[]
    data[] ---> Temporary array to store a current combination
    i      ---> Index of current element in vertices[]
*/
void combineUntil(int index, vector<int>& data, int i)
{
    if (index == C)
    {
        for (int j = 0; j < C; ++j)
            N->seeds[qtd*C + j] = data[j];
        qtd++;
        return;
    }
 
    if (i >= SIZE) return;

    data[index] = i;
    combineUntil(index+1, data, i+1);
    combineUntil(index, data, i+1);
}
//-----------------------------------------------------------------------------
void combine()
{
    vector<int> data(C);
    combineUntil(0, data, 0);
}
//-----------------------------------------------------------------------------
/*
    Defines the number of combinations.
    */
void sizeDefinitions()
{
    for (int i = 6; i <= MAX; ++i)
    {
        int resp = 1;
        for (int j = i-3; j <= i; ++j) resp *= j;
        resp /= 24;
        bib[i-1] = resp;
    }
}
//-----------------------------------------------------------------------------
void initialize()
{
    for (int i = 0; i < SIZE; ++i)
    {
        for (int j = i+1; j < SIZE; ++j)
            R[i*SIZE + j] = R[j*SIZE + i] = -1;
        R[i*SIZE + i] = -1;
    }
}
//-----------------------------------------------------------------------------
void readInput()
{
    int x;
    scanf("%d", &SIZE);
    PERM = bib[SIZE-1];

    N = (Node*)malloc(sizeof(Node));
    N->sz = SIZE;

    for (int i = 0; i < SIZE; ++i)
    {
        for (int j = i+1; j < SIZE; ++j)
        {
            scanf("%d", &x);
            N->graph[i*SIZE + j] = x;
            N->graph[j*SIZE + i] = x;
        }
        N->graph[i*SIZE + i] = -1;
    }
}
//-----------------------------------------------------------------------------
int main(int argv, char** argc)
{
    sizeDefinitions();
    // Reads the input, which is given by the size of a graph and its weighted
    // edges. The given graph should be a complete graph.
    readInput();
    initialize();
    // Generate 4-clique seeds, given the number of vertices
    combine();

    if (argv == 2) hipSetDevice(atoi(argc[1]));
    else if (argv == 3)
    {
        GPU_CNT = atoi(argc[2]);
        int d;
        hipGetDeviceCount(&d);
        if (GPU_CNT > d) GPU_CNT = d;
    }

    double start = getTime();
    int respMax = prepare();
    double stop = getTime();

    //reconstruct the graph given the regions of the graph
    for (int i = 0; i < 2 * SIZE; ++i)
    {
        int va = F[i * 3], vb = F[i * 3 + 1], vc = F[i * 3 + 2];
        if (va == vb && vb == vc) continue;
        R[va*SIZE + vb] = R[vb*SIZE + va] = N->graph[va*SIZE + vb];
        R[va*SIZE + vc] = R[vc*SIZE + va] = N->graph[va*SIZE + vc];
        R[vb*SIZE + vc] = R[vc*SIZE + vb] = N->graph[vb*SIZE + vc];
    }

    printf("Printing generated graph:\n");
    for (int i = 0; i < SIZE; ++i)
    {
        for (int j = i + 1; j < SIZE; ++j)
            printf("%d ", R[i*SIZE + j]);
        printf("\n");
    }

    printElapsedTime(start, stop);
    cout << "Maximum weight found: " << respMax << endl;
    free(N);

    return 0;
}
#include "hip/hip_runtime.h"
#include <cmath>
#include <cstdio>
#include <cstdlib>

#define C 4
#define THREADS 1024 // 2^10
#define MAX 85
#define MAXS MAX* MAX
#define COMB_MAX (MAX * (MAX - 1) * (MAX - 2) * (MAX - 3)) / 24

#define gpuErrChk(ans)                        \
    {                                         \
        gpuAssert((ans), __FILE__, __LINE__); \
    }
inline void gpuAssert(hipError_t code, char* file, int line, bool abort = true)
{
    if (code != hipSuccess) {
        fprintf(stderr, "GPUassert: %s %s %d\n", hipGetErrorString(code),
            file, line);
        if (abort)
            getchar();
    }
}

using namespace std;

struct Node {
    int sz, qtd;
    int graph[MAXS], TMP[6 * MAX], seeds[COMB_MAX * C];
};

struct Params {
    int faces, count, tmpMax;
    int F[6 * MAX], V[MAX];
};

/*
    SIZE        ---> Number of vertices
    C           ---> Size of the combination (Size of a seed clique)
    faces       ---> Quantity of triangular faces
    qtd         ---> Number of possible 4-cliques
    T           ---> Output graph for an instance
    R           ---> Output graph for an possible optimal solution
    F           ---> List containing triangular faces of an instance
    seeds       ---> Combinations of possible starting 4-cliques
    graph       ---> The graph itself
*/
double start, stop;
int R[MAX][MAX], F[8 * MAX], bib[MAX];
int SIZE, BLOCKS, COMB, qtd = 0;

Node* N;

//-----------------------------------------------------------------------------
// Mac OSX
#ifdef __MACH__
#include <mach/clock.h>
#include <mach/mach.h>
#endif

/*
    Prints elapsed time.
    */
void printElapsedTime(double start, double stop)
{
    double elapsed = stop - start;
    printf("Elapsed time: %.3lfs.\n", elapsed);
}
//-----------------------------------------------------------------------------
/*  
    Gets clock time.
    */
void current_utc_time(struct timespec* ts)
{
#ifdef __MACH__ // OS X does not have clock_gettime, use clock_get_time
    clock_serv_t cclock;
    mach_timespec_t mts;
    host_get_clock_service(mach_host_self(), CALENDAR_CLOCK, &cclock);
    clock_get_time(cclock, &mts);
    mach_port_deallocate(mach_task_self(), cclock);
    ts->tv_sec = mts.tv_sec;
    ts->tv_nsec = mts.tv_nsec;
#else
    clock_gettime(CLOCK_REALTIME, ts);
#endif
}
//-----------------------------------------------------------------------------
double getTime()
{
    timespec ts;
    current_utc_time(&ts);
    return double(ts.tv_sec) + double(ts.tv_nsec) / 1e9;
}
//-----------------------------------------------------------------------------
/*
    t   ---> thread index
    Generates a list of vertices which are not on the initial planar graph.
    */
__device__ void generateVertexList(Node* devN, Params* devP, int t)
{
    int sz = devN->sz;
    int va = devN->seeds[t * 4], vb = devN->seeds[t * 4 + 1],
        vc = devN->seeds[t * 4 + 2], vd = devN->seeds[t * 4 + 3];
    for (int i = 0; i < sz; ++i) {
        if (i == va || i == vb || i == vc || i == vd)
            devP[t].V[i] = -1;
        else
            devP[t].V[i] = i;
    }
}
//-----------------------------------------------------------------------------
/*
    t   ---> thread index
    Returns the initial solution weight for the planar graph and
    initializes necessary structures, such as the edges indexes,
    and defines which edges belongs to a face.
    */
__device__ void generateFaceList(Node* devN, Params* devP, int t)
{
    int resp = 0, sz = devN->sz;
    int va = devN->seeds[t * 4], vb = devN->seeds[t * 4 + 1],
        vc = devN->seeds[t * 4 + 2], vd = devN->seeds[t * 4 + 3];

    // Generate the first triangle of the output graph
    devP[t].F[devP[t].faces * 3] = va,
                              devP[t].F[devP[t].faces * 3 + 1] = vb,
                              devP[t].F[(devP[t].faces++) * 3 + 2] = vc;
    resp = devN->graph[va * sz + vb] + devN->graph[va * sz + vc] + devN->graph[vb * sz + vc];

    // Generate the next 3 possible faces
    devP[t].F[devP[t].faces * 3] = va, devP[t].F[devP[t].faces * 3 + 1] = vb,
                              devP[t].F[(devP[t].faces++) * 3 + 2] = vd;
    devP[t].F[devP[t].faces * 3] = va, devP[t].F[devP[t].faces * 3 + 1] = vc,
                              devP[t].F[(devP[t].faces++) * 3 + 2] = vd;
    devP[t].F[devP[t].faces * 3] = vb, devP[t].F[devP[t].faces * 3 + 1] = vc,
                              devP[t].F[(devP[t].faces++) * 3 + 2] = vd;
    resp += devN->graph[va * sz + vd] + devN->graph[vb * sz + vd] + devN->graph[vc * sz + vd];
    devP[t].tmpMax = resp;
}
//-----------------------------------------------------------------------------
/*
    Inserts a new vertex, 3 new triangular faces
    and removes the face from the list.
    */
__device__ int faceDimple(Node* devN, Params* devP, int new_vertex, int f, int t)
{
    // Remove the chosen face and insert a new one
    int va = devP[t].F[f * 3],
        vb = devP[t].F[f * 3 + 1],
        vc = devP[t].F[f * 3 + 2];

    devP[t].F[f * 3] = new_vertex,
                  devP[t].F[f * 3 + 1] = va, devP[t].F[f * 3 + 2] = vb;
    // Insert the other two possible faces
    devP[t].F[devP[t].faces * 3] = new_vertex,
                              devP[t].F[devP[t].faces * 3 + 1] = va,
                              devP[t].F[(devP[t].faces++) * 3 + 2] = vc;
    devP[t].F[devP[t].faces * 3] = new_vertex,
                              devP[t].F[devP[t].faces * 3 + 1] = vb,
                              devP[t].F[(devP[t].faces++) * 3 + 2] = vc;

    int sz = devN->sz;
    int resp = devN->graph[va * sz + new_vertex] + devN->graph[vb * sz + new_vertex]
        + devN->graph[vc * sz + new_vertex];

    return resp;
}
//-----------------------------------------------------------------------------
/*
    Returns the vertex having the maximum gain
    inserting within a face.
    */
__device__ int maxGainFace(Node* devN, Params* devP, int* f, int t)
{
    int sz = devN->sz;
    int gain = -1, vertex = -1;

    int faces = devP[t].faces;
    // Iterate through the remaining vertices
    for (int new_vertex = 0; new_vertex < sz; ++new_vertex) {
        if (devP[t].V[new_vertex] == -1)
            continue;
        // Test the dimple on each face
        for (int i = 0; i < faces; ++i) {
            int va = devP[t].F[i * 3], vb = devP[t].F[i * 3 + 1], vc = devP[t].F[i * 3 + 2];
            int tmpGain = devN->graph[va * sz + new_vertex] + devN->graph[vb * sz + new_vertex]
                + devN->graph[vc * sz + new_vertex];
            if (tmpGain > gain) {
                gain = tmpGain;
                *f = i;
                vertex = new_vertex;
            }
        }
    }
    return vertex;
}
//-----------------------------------------------------------------------------
__device__ void dimpling(Node* devN, Params* devP, int t)
{
    while (devP[t].count) {
        int f = -1;
        int vertex = maxGainFace(devN, devP, &f, t);
        devP[t].V[vertex] = -1;
        devP[t].tmpMax += faceDimple(devN, devP, vertex, f, t);
        devP[t].count--;
    }
}
//-----------------------------------------------------------------------------
__device__ void initializeDevice(Params* devP, int sz, int t)
{
    devP[t].faces = 0;
    devP[t].tmpMax = -1;
    devP[t].count = sz - 4;
}
//-----------------------------------------------------------------------------
__global__ void solve(Node* devN, Params* devP, int* respMax, int* idx)
{
    int x = blockDim.x * blockIdx.x + threadIdx.x;
    int sz = devN->sz;
    int comb = devN->qtd;
    __syncthreads();

    if (x < comb) {
        initializeDevice(devP, devN->sz, x);
        generateVertexList(devN, devP, x);
        generateFaceList(devN, devP, x);
        dimpling(devN, devP, x);

        __syncthreads();
        atomicMax(respMax, devP[x].tmpMax);
        if (devP[x].tmpMax == *respMax)
            *idx = x;
        __syncthreads();
    }
}
//-----------------------------------------------------------------------------
int prepare()
{
    int resp = 0, idx = 0, *tmpResp, *tmpIdx;
    gpuErrChk(hipMalloc((void**)&tmpResp, sizeof(int)));
    gpuErrChk(hipMalloc((void**)&tmpIdx, sizeof(int)));
    gpuErrChk(hipMemcpy(tmpResp, &resp, sizeof(int), hipMemcpyHostToDevice));
    gpuErrChk(hipMemcpy(tmpIdx, &idx, sizeof(int), hipMemcpyHostToDevice));

    Node* devN;
    Params* devP;

    size_t sz = COMB * sizeof(Params);
    gpuErrChk(hipMalloc((void**)&devP, sz));
    gpuErrChk(hipMalloc((void**)&devN, sizeof(Node)));
    gpuErrChk(hipMemcpy(devN, N, sizeof(Node), hipMemcpyHostToDevice));

    dim3 blocks(BLOCKS, 1);
    dim3 threads(THREADS, 1);

    solve<<<blocks, threads>>>(devN, devP, tmpResp, tmpIdx);
    gpuErrChk(hipDeviceSynchronize());

    gpuErrChk(hipMemcpy(&resp, tmpResp, sizeof(int), hipMemcpyDeviceToHost));
    gpuErrChk(hipMemcpy(&idx, tmpIdx, sizeof(int), hipMemcpyDeviceToHost));
    gpuErrChk(hipMemcpy(&F, devP[idx].F, (6 * MAX) * sizeof(int), hipMemcpyDeviceToHost));

    gpuErrChk(hipFree(devN));
    gpuErrChk(hipFree(devP));
    return resp;
}
//-----------------------------------------------------------------------------
/*
    C      ---> Size of the combination
    index  ---> Current index in data[]
    data[] ---> Temporary array to store a current combination
    i      ---> Index of current element in vertices[]
*/
void combineUntil(int index, int* data, int i)
{
    if (index == C) {
        for (int j = 0; j < C; ++j)
            N->seeds[qtd * C + j] = data[j];
        qtd++;
        return;
    }

    if (i >= SIZE)
        return;

    data[index] = i;
    combineUntil(index + 1, data, i + 1);
    combineUntil(index, data, i + 1);
}
//-----------------------------------------------------------------------------
void combine()
{
    int data[C];
    combineUntil(0, data, 0);
}
//-----------------------------------------------------------------------------
/*
    Defines the number of combinations.
    */
void sizeDefinitions()
{
    for (int i = 4; i <= MAX; ++i) {
        int resp = 1;
        for (int j = i - 3; j <= i; ++j)
            resp *= j;
        resp /= 24;
        bib[i - 1] = resp;
    }
}
//-----------------------------------------------------------------------------
void initialize()
{
    for (int i = 0; i < SIZE; ++i) {
        for (int j = i + 1; j < SIZE; ++j)
            R[i][j] = R[j][i] = -1;
        R[i][i] = -1;
    }
}
//-----------------------------------------------------------------------------
void readInput()
{
    int x;
    scanf("%d", &SIZE);
    COMB = bib[SIZE - 1];
    BLOCKS = COMB / THREADS + 1;

    N = (Node*)malloc(sizeof(Node));
    N->sz = SIZE;
    N->qtd = COMB;

    for (int i = 0; i < SIZE; ++i) {
        for (int j = i + 1; j < SIZE; ++j) {
            scanf("%d", &x);
            N->graph[i * SIZE + j] = x;
            N->graph[j * SIZE + i] = x;
        }
        N->graph[i * SIZE + i] = -1;
    }
}
//-----------------------------------------------------------------------------
int main(int argv, char** argc)
{
    sizeDefinitions();
    // Reads the input, which is given by the size of a graph and its weighted
    // edges. The given graph should be a complete graph.
    readInput();
    initialize();
    // Generate 4-clique seeds, given the number of vertices
    combine();

    start = getTime();
    int respMax = prepare();
    stop = getTime();

    for (int i = 0; i < 2 * SIZE; ++i) {
        int va = F[i * 3], vb = F[i * 3 + 1], vc = F[i * 3 + 2];
        if (va == vb && vb == vc)
            continue;
        R[va][vb] = R[vb][va] = N->graph[va * SIZE + vb];
        R[va][vc] = R[vc][va] = N->graph[va * SIZE + vc];
        R[vb][vc] = R[vc][vb] = N->graph[vb * SIZE + vc];
    }

    printf("Printing generated graph:\n");
    for (int i = 0; i < SIZE; ++i) {
        for (int j = i + 1; j < SIZE; ++j)
            printf("%d ", R[i][j]);
        printf("\n");
    }

    printElapsedTime(start, stop);
    printf("Maximum weight found: %d\n", respMax);
    free(N);

    return 0;
}
